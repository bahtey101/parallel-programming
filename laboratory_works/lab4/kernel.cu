
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <malloc.h>
#include <stdio.h>
#include <stdlib.h>

#define NMAX 6000000
#define ITERATIONS 20

__global__ void addKernel(int* a, int* b, int* s, unsigned int size) {
    int gridSize = blockDim.x * gridDim.x;
    int start = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = start; i < size; i += gridSize) {
        s[i] = a[i] + b[i];
    }
}

int main(int argc, char* argv[]) {
    int GRID_DIM  = 128;
    int BLOCK_DIM = 1024;

    if (argc > 2) {
       GRID_DIM  = atoi(argv[1]);
       BLOCK_DIM = atoi(argv[2]);
   }

    int n2i = NMAX * sizeof(int);
    // Выделение памяти на хосте
    int* a = (int*)calloc(NMAX, sizeof(int));
    int* b = (int*)calloc(NMAX, sizeof(int));
    int* s = (int*)calloc(NMAX, sizeof(int));

    // Инициализация массивов
    for (int i = 0; i < NMAX; i++) {
        a[i] = 1;
        b[i] = 2;
    }

    // Выделение памяти на устройстве
    int* adev = NULL;
    hipError_t cuerr = hipMalloc((void**)&adev, n2i);
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, "Cannot allocate device array for a: %s\n",
            hipGetErrorString(cuerr));
        return 0;
    }

    int* bdev = NULL;
    cuerr = hipMalloc((void**)&bdev, n2i);
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, "Cannot allocate device array for b: %s\n",
            hipGetErrorString(cuerr));
        return 0;
    }

    int* sdev = NULL;
    cuerr = hipMalloc((void**)&sdev, n2i);
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, "Cannot allocate device array for s: %s\n",
            hipGetErrorString(cuerr));
        return 0;
    }

    // Создание обработчиков событий
    hipEvent_t start, stop;
    float seqTime = 0.0f;
    float gpuTime = 0.0f;
    float trTime  = 0.0f;
    cuerr = hipEventCreate(&start);
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, "Cannot create CUDA start event: %s\n",
            hipGetErrorString(cuerr));
        return 0;
    }

    cuerr = hipEventCreate(&stop);
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, "Cannot create CUDA end event: %s\n",
            hipGetErrorString(cuerr));
        return 0;
    }

    for(int cntr = 0; cntr < ITERATIONS; ++cntr) {
        // Замер последовательного алгоритма
        float seqTimetmp = 0.0f;

        cuerr = hipEventRecord(start, 0);
        if (cuerr != hipSuccess) 
        {
            fprintf(stderr, "Cannot record CUDA event: %s\n",
                hipGetErrorString(cuerr));
            return 0;
        }

        for (int i = 0; i < NMAX; ++i) {
            s[i] = a[i] + b[i];
        }

        cuerr = hipEventRecord(stop, 0);
        if (cuerr != hipSuccess)
        {
            fprintf(stderr, "Cannot record CUDA event: %s\n",
                hipGetErrorString(cuerr));
            return 0;
        }

        cuerr = hipDeviceSynchronize();
        if (cuerr != hipSuccess)
        {
            fprintf(stderr, "Cannot synchronize CUDA kernel: %s\n",
                hipGetErrorString(cuerr));
            return 0;
        }

        cuerr = hipEventElapsedTime(&seqTimetmp, start, stop);
        if (cuerr != hipSuccess)
        {
            fprintf(stderr, "1Cannot calculate elapsed time: %s\n",
                hipGetErrorString(cuerr));
            return 0;
        }



        // Параллельный алгоритим
        // Замер времени передачи данных на видеокарту
        float to_device_tmp = 0.0f;
        cuerr = hipEventRecord(start, 0);
        if (cuerr != hipSuccess) 
        {
            fprintf(stderr, "Cannot record CUDA event: %s\n",
                hipGetErrorString(cuerr));
            return 0;
        }   

        cuerr = hipMemcpy(adev, a, n2i, hipMemcpyHostToDevice);
        if (cuerr != hipSuccess)
        {
            fprintf(stderr, "Cannot copy a array from host to device: %s\n",
                hipGetErrorString(cuerr));
            return 0;
        }

        cuerr = hipMemcpy(bdev, b, n2i, hipMemcpyHostToDevice);
        if (cuerr != hipSuccess)
        {
            fprintf(stderr, "Cannot copy b array from host to device: %s\n",
                hipGetErrorString(cuerr));
            return 0;
        }

        cuerr = hipEventRecord(stop, 0);
        if (cuerr != hipSuccess)
        {
            fprintf(stderr, "Cannot record CUDA event: %s\n",
                hipGetErrorString(cuerr));
            return 0;
        }

        cuerr = hipDeviceSynchronize();
        if (cuerr != hipSuccess)
        {
            fprintf(stderr, "Cannot synchronize CUDA kernel: %s\n",
                hipGetErrorString(cuerr));
            return 0;
        }

        cuerr = hipEventElapsedTime(&to_device_tmp, start, stop);
        if (cuerr != hipSuccess)
        {
            fprintf(stderr, "2Cannot calculate elapsed time: %s\n",
                hipGetErrorString(cuerr));
            return 0;
        }


        
        // Замер времени выполнения ядра
        float gpuTimetmp = 0.0f;
        cuerr = hipEventRecord(start, 0);
        if (cuerr != hipSuccess) 
        {
            fprintf(stderr, "Cannot record CUDA event: %s\n",
                hipGetErrorString(cuerr));
            return 0;
        }

        addKernel <<< GRID_DIM, BLOCK_DIM >>> (adev, bdev, sdev, NMAX);

        cuerr = hipGetLastError();
        if (cuerr != hipSuccess)
        {
            fprintf(stderr, "Cannot launch CUDA kernel: %s\n",
                hipGetErrorString(cuerr));
            return 0;
        }

        cuerr = hipEventRecord(stop, 0);
        if (cuerr != hipSuccess) {
            fprintf(stderr, "Cannot record stop CUDA event: %s\n",
                hipGetErrorString(cuerr));
            return 0;
        }

        cuerr = hipDeviceSynchronize();
        if (cuerr != hipSuccess)
        {
            fprintf(stderr, "3Cannot synchronize CUDA kernel: %s\n",
                hipGetErrorString(cuerr));
            return 0;
        }

        cuerr = hipEventElapsedTime(&gpuTimetmp, start, stop);
        if (cuerr != hipSuccess) {
            fprintf(stderr, "Cannot calculate elapsed time: %s\n", 
                hipGetErrorString(cuerr));
            return 0;
        }



        // Замер времени передачи данных с видеокарты в ОЗУ
        float to_host_tmp = 0.0f;
        cuerr = hipEventRecord(start, 0);
        if (cuerr != hipSuccess) 
        {
            fprintf(stderr, "Cannot record CUDA event: %s\n",
                hipGetErrorString(cuerr));
            return 0;
        }

        cuerr = hipMemcpy(s, sdev, n2i, hipMemcpyDeviceToHost);
        if (cuerr != hipSuccess)
        {
            fprintf(stderr, "Cannot copy c array from device to host: %s\n",
                hipGetErrorString(cuerr));
            return 0;
        }

        cuerr = hipEventRecord(stop, 0);
        if (cuerr != hipSuccess) {
            fprintf(stderr, "Cannot record stop CUDA event: %s\n",
                hipGetErrorString(cuerr));
            return 0;
        }

        cuerr = hipDeviceSynchronize();
        if (cuerr != hipSuccess)
        {
            fprintf(stderr, "Cannot synchronize CUDA kernel: %s\n",
                hipGetErrorString(cuerr));
            return 0;
        }

        cuerr = hipEventElapsedTime(&to_host_tmp, start, stop);
        if (cuerr != hipSuccess) {
            fprintf(stderr, "4Cannot calculate elapsed time: %s\n", 
                hipGetErrorString(cuerr));
            return 0;
        }

        seqTime += seqTimetmp / ITERATIONS;
        gpuTime += gpuTimetmp / ITERATIONS;
        trTime  += (to_device_tmp + to_host_tmp) / ITERATIONS; 
    }

    printf("----------- CUDA -----------\n");
    printf("  DATA TYPE        : %s\n", "int");
    printf("  VECTORS SIZE     : %d\n", NMAX);
    printf("  VECTORS NUMBER   : %d\n", 2);
    printf("  GRID DIM         : %d\n", GRID_DIM);
    printf("  BLOCK DIM        : %d\n", BLOCK_DIM);
    printf("----------------------------\n\n");

    printf(" Sequental time    : %f ms\n",   seqTime);
    printf(" Time to send(tr)  : %f ms\n",   trTime);
    printf(" CUDA time of work : %f ms\n\n", gpuTime);

    printf(" CUDA acceleration w/i send : %f\n",   seqTime / (gpuTime + trTime));
    printf(" CUDA acceleration w/o send : %f\n\n", seqTime / gpuTime);

    printf(" VECTOR <S>\n");
    for (int i = 0; i < 3; ++i) {
        printf(" s[%d] = %d\n", i, s[i]);
    }
    printf("    ...\n");
    for (int i = -3; i < 0; ++i) {
        printf(" s[%d] = %d\n", NMAX + i, s[NMAX + i]);
    }

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(adev);
    hipFree(bdev);
    hipFree(sdev);
    free(a);
    free(b);
    free(s);

    return 0;
}

